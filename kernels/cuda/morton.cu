#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <glm/glm.hpp>

#include "types/morton.hpp"

namespace gpu {

__device__ __forceinline__ unsigned int morton3D_SplitBy3bits(
    const unsigned int a) {
  unsigned int x = ((unsigned int)a) & 0x000003ff;
  x = (x | x << 16) & 0x30000ff;
  x = (x | x << 8) & 0x0300f00f;
  x = (x | x << 4) & 0x30c30c3;
  x = (x | x << 2) & 0x9249249;
  return x;
}

__device__ __forceinline__ unsigned int m3D_e_magicbits(const unsigned int x,
                                                        const unsigned int y,
                                                        const unsigned int z) {
  return morton3D_SplitBy3bits(x) | (morton3D_SplitBy3bits(y) << 1) |
         (morton3D_SplitBy3bits(z) << 2);
}

[[nodiscard]] __device__ __forceinline__ unsigned int xyz_to_morton32(
    const glm::vec4& xyz, const float min_coord, const float range) {
  constexpr auto bit_scale = 1024;
  const auto i = static_cast<uint32_t>((xyz.x - min_coord) / range * bit_scale);
  const auto j = static_cast<uint32_t>((xyz.y - min_coord) / range * bit_scale);
  const auto k = static_cast<uint32_t>((xyz.z - min_coord) / range * bit_scale);
  return m3D_e_magicbits(i, j, k);
}

__global__ void k_ComputeMorton(const glm::vec4* d_xyz,
                                unsigned int* d_morton,
                                const int n,
                                const float min_coord,
                                const float range) {
  const auto idx = threadIdx.x + blockDim.x * blockIdx.x;
  const auto stride = blockDim.x * gridDim.x;

  for (auto i = idx; i < n; i += stride)
    d_morton[i] = xyz_to_morton32(d_xyz[i], min_coord, range);
}

// assume n = 1024
// k_ComputeMorton<<<4, 256>>>(u_data);
//   Do task...
// 
// k_ComputeMorton<<<1, 256>>>(u_data);
// for(i = 0; i < 4; ++i) {
//  Do task...
// }
//  


void Dispatch_ComputeMortonCode_With(const glm::vec4* data,
                                     MortonT* morton_keys,
                                     size_t n,
                                     float min_coord,
                                     float range,
                                     // gpu thing
                                     int logical_num_blocks) {
  constexpr auto block_size = 768;
  k_ComputeMorton<<<logical_num_blocks, block_size>>>(
      data, morton_keys, n, min_coord, range);
}

}  // namespace gpu

// void k_ComputeMortonCode(const glm::vec4* data,
//                          unsigned int* morton_keys,
//                          const size_t n,
//                          const float min_coord,
//                          const float range) {
//   gpu::k_ComputeMorton<<<1, 1>>>(data, morton_keys, n, min_coord, range);
// }
