
#include <hip/hip_runtime.h>


namespace gpu {

__global__ void k_CountUnique(unsigned int* keys,
                              int* num_unique_out,
                              const int n) {
  if (const auto tid = threadIdx.x + blockIdx.x * blockDim.x; tid == 0) {
    if (n == 0) {
      *num_unique_out = 0;
      return;
    }

    auto j = 0;
    for (auto i = 1; i < n; ++i) {
      if (keys[i] != keys[j]) {
        ++j;
        keys[j] = keys[i];
      }
    }

    *num_unique_out = j + 1;
  }
}

}  // namespace gpu