#include <hip/hip_runtime.h>

#include <hipcub/hipcub.hpp>
#include <cuda/common/helper_cuda.hpp>

namespace gpu {
void k_CubRadixSort(unsigned int *keys, unsigned int *keys_alt, int n) {
  size_t temp_storage_bytes = 0;
  void *d_temp_storage = nullptr;

  hipcub::DeviceRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, keys, keys_alt, n);

  checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));

  hipcub::DeviceRadixSort::SortKeys(
      d_temp_storage, temp_storage_bytes, keys, keys_alt, n);

  checkCudaErrors(hipFree(d_temp_storage));
}
}  // namespace gpu