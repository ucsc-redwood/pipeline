#include <benchmark/benchmark.h>

#include <glm/glm.hpp>

#include "config.hpp"
#include "cuda_bench_helper.cuh"
#include "kernels/init.hpp"
#include "kernels/morton.hpp"

namespace bm = benchmark;

static void BM_Morton32(bm::State& st) {
  const auto n_blocks = st.range(0);

  auto d_xyz = AllocateDevice<glm::vec4>(kN);
  auto d_morton = AllocateDevice<unsigned int>(kN);

  gpu::k_InitRandomVec4(d_xyz, kN, kMin, kRange, kRandomSeed);
  BENCH_CUDA_TRY(hipDeviceSynchronize());

  for (auto _ : st) {
    cuda_event_timer timer(st, true);

    gpu::Dispatch_ComputeMortonCode_With(
        d_xyz, d_morton, kN, kMin, kRange, n_blocks);
  }

  Free(d_xyz);
  Free(d_morton);
}

BENCHMARK(BM_Morton32)
    ->RangeMultiplier(2)
    ->Range(1, 1 << 10)
    ->UseManualTime()
    ->Unit(bm::kMillisecond);

BENCHMARK_MAIN();
