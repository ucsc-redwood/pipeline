#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>
#include <spdlog/spdlog.h>

#include <CLI/CLI.hpp>
#include <glm/glm.hpp>

#include "cuda/common/helper_cuda.hpp"
#include "cuda/kernels/all.cuh"
#include "shared/types.h"

template <typename T>
[[nodiscard]] T* AllocManaged(const int n) {
  T* ptr;
  checkCudaErrors(hipMallocManaged(&ptr, n * sizeof(T)));
  return ptr;
}

template <typename T>
void AllocManaged(T** ptr, const int n) {
  checkCudaErrors(hipMallocManaged(ptr, n * sizeof(T)));
}

const int radix = 256;
const int radixPasses = 4;
const int partitionSize = 7680;
const int globalHistPartitionSize = 65536;
const int globalHistThreads = 128;
const int binningThreads = 512;  // 2080 super seems to really like 512

constexpr int binningThreadblocks(const int size) {
  return (size + partitionSize - 1) / partitionSize;
}

constexpr int globalHistThreadblocks(const int size) {
  return (size + globalHistPartitionSize - 1) / globalHistPartitionSize;
}

void InitMemory(unsigned int* u_index,
                unsigned int* globalHistogram,
                unsigned int* firstPassHistogram,
                unsigned int* secPassHistogram,
                unsigned int* thirdPassHistogram,
                unsigned int* fourthPassHistogram,
                const int n) {
  hipMemset(u_index, 0, radixPasses * sizeof(unsigned int));
  hipMemset(globalHistogram, 0, radix * radixPasses * sizeof(unsigned int));
  hipMemset(firstPassHistogram,
             0,
             radix * binningThreadblocks(n) * sizeof(unsigned int));
  hipMemset(secPassHistogram,
             0,
             radix * binningThreadblocks(n) * sizeof(unsigned int));
  hipMemset(thirdPassHistogram,
             0,
             radix * binningThreadblocks(n) * sizeof(unsigned int));
  hipMemset(fourthPassHistogram,
             0,
             radix * binningThreadblocks(n) * sizeof(unsigned int));
}

void DispatchSortKernels(OneSweepData<4>& one_sweep, const int n) {
  spdlog::info("dispatching radix sort... with {} blocks",
               globalHistThreadblocks(n));

  gpu::k_GlobalHistogram<<<globalHistThreadblocks(n), globalHistThreads>>>(
      one_sweep.u_sort, one_sweep.u_global_histogram, n);

  spdlog::info("dispatching k_DigitBinning... with {} blocks",
               binningThreadblocks(n));

  gpu::k_DigitBinning<<<binningThreadblocks(n), binningThreads>>>(
      one_sweep.u_global_histogram,
      one_sweep.u_sort,
      one_sweep.u_sort_alt,
      one_sweep.u_pass_histograms[0],
      one_sweep.u_index,
      n,
      0);

  gpu::k_DigitBinning<<<binningThreadblocks(n), binningThreads>>>(
      one_sweep.u_global_histogram,
      one_sweep.u_sort_alt,
      one_sweep.u_sort,
      one_sweep.u_pass_histograms[1],
      one_sweep.u_index,
      n,
      8);

  gpu::k_DigitBinning<<<binningThreadblocks(n), binningThreads>>>(
      one_sweep.u_global_histogram,
      one_sweep.u_sort,
      one_sweep.u_sort_alt,
      one_sweep.u_pass_histograms[2],
      one_sweep.u_index,
      n,
      16);

  gpu::k_DigitBinning<<<binningThreadblocks(n), binningThreads>>>(
      one_sweep.u_global_histogram,
      one_sweep.u_sort_alt,
      one_sweep.u_sort,
      one_sweep.u_pass_histograms[3],
      one_sweep.u_index,
      n,
      24);
}

int main(const int argc, const char** argv) {
  int n = 10'000'000;
  // int n = size;
  int n_threads = 4;
  int my_num_blocks = 64;

  CLI::App app{"Multi-threaded sorting benchmark"};

  app.add_option("-n,--n", n, "Number of elements to sort")
      ->check(CLI::PositiveNumber);

  app.add_option("-t,--threads", n_threads, "Number of threads to use")
      ->check(CLI::Range(1, 48));

  app.add_option("-b,--blocks", my_num_blocks, "Number of blocks to use")
      ->check(CLI::PositiveNumber);

  CLI11_PARSE(app, argc, argv)

  spdlog::info("n = {}", n);
  spdlog::info("n_threads = {}", n_threads);
  spdlog::info("my_num_blocks = {}", my_num_blocks);

  omp_set_num_threads(n_threads);

  // ---------------------------------------------------------------------------

  constexpr auto min = 0.0f;
  constexpr auto max = 1024.0f;
  constexpr auto range = max - min;

  auto u_data = AllocManaged<glm::vec4>(n);

  OneSweepData<radixPasses> one_sweep;
  AllocManaged(&one_sweep.u_sort, n);
  AllocManaged(&one_sweep.u_sort_alt, n);
  AllocManaged(&one_sweep.u_global_histogram, radix * radixPasses);
  AllocManaged(&one_sweep.u_index, radixPasses);
  for (int i = 0; i < radixPasses; ++i) {
    AllocManaged(&one_sweep.u_pass_histograms[i],
                 radix * binningThreadblocks(n));
  }

  {
    constexpr auto num_threads = 768;
    constexpr auto seed = 114514;
    const auto grid_size = (n + num_threads - 1) / num_threads;
    gpu::k_InitRandomVec4<<<grid_size, num_threads>>>(
        u_data, n, min, range, seed);
    checkCudaErrors(hipDeviceSynchronize());
  }

  // peek 10 elements
  for (int i = 0; i < 10; ++i) {
    spdlog::debug("u_data[{}] = ({}, {}, {}, {})",
                  i,
                  u_data[i].x,
                  u_data[i].y,
                  u_data[i].z,
                  u_data[i].w);
  }

  {
    constexpr auto num_threads = 768;
    gpu::k_ComputeMortonCode<<<my_num_blocks, num_threads>>>(
        u_data, one_sweep.u_sort, n, min, range);
    checkCudaErrors(hipDeviceSynchronize());
  }

  // peek 10 elements
  for (int i = 0; i < 10; ++i) {
    spdlog::debug("one_sweep.u_sort[{}] = {}", i, one_sweep.u_sort[i]);
  }

  // Sorting kernels
  DispatchSortKernels(one_sweep, n);
  checkCudaErrors(hipDeviceSynchronize());

  spdlog::info("Done Sorting!");

  // peek 10 elements
  for (int i = 0; i < 10; ++i) {
    spdlog::info("one_sweep.u_sort[{}] = {}", i, one_sweep.u_sort[i]);
  }

  const auto is_sorted = std::is_sorted(one_sweep.u_sort, one_sweep.u_sort + n);
  std::cout << "is_sorted = " << std::boolalpha << is_sorted << '\n';

  // ---------------------------------------------------------------------------
  // TMP

  int* num_unique_out;
  checkCudaErrors(hipMallocManaged(&num_unique_out, sizeof(int)));

  unsigned int* u_temp_sort;
  checkCudaErrors(hipMallocManaged(&u_temp_sort, n * sizeof(unsigned int)));
  std::copy(one_sweep.u_sort, one_sweep.u_sort + n, u_temp_sort);

  gpu::k_CountUnique<<<1, 1>>>(u_temp_sort, num_unique_out, n);
  checkCudaErrors(hipDeviceSynchronize());

  spdlog::info("num_unique_out = {}", *num_unique_out);

  checkCudaErrors(hipFree(num_unique_out));
  checkCudaErrors(hipFree(u_temp_sort));

  const auto it = std::unique(one_sweep.u_sort, one_sweep.u_sort + n);
  const auto num_unique = std::distance(one_sweep.u_sort, it);

  spdlog::info("num_unique = {}", num_unique);

  assert(num_unique == *num_unique_out);

  // ---------------------------------------------------------------------------

  RadixTreeData tree;
  tree.n_nodes = num_unique - 1;
  AllocManaged(&tree.prefixN, tree.n_nodes);
  AllocManaged(&tree.hasLeafLeft, tree.n_nodes);
  AllocManaged(&tree.hasLeafRight, tree.n_nodes);
  AllocManaged(&tree.leftChild, tree.n_nodes);
  AllocManaged(&tree.parent, tree.n_nodes);

  gpu::k_BuildRadixTree<<<my_num_blocks, 768>>>(num_unique,
                                                one_sweep.u_sort,
                                                tree.prefixN,
                                                tree.hasLeafLeft,
                                                tree.hasLeafRight,
                                                tree.leftChild,
                                                tree.parent);
  checkCudaErrors(hipDeviceSynchronize());

  // peek 10 nodes
  for (int i = 0; i < 10; ++i) {
    spdlog::trace(
        "tree.prefixN[{}] = {}, tree.hasLeafLeft[{}] = {}, "
        "tree.hasLeafRight[{}] = {}, tree.leftChild[{}] = {}, "
        "tree.parent[{}] = {}",
        i,
        tree.prefixN[i],
        i,
        tree.hasLeafLeft[i],
        i,
        tree.hasLeafRight[i],
        i,
        tree.leftChild[i],
        i,
        tree.parent[i]);
  }

  // ---------------------------------------------------------------------------
  // Cleanup

  checkCudaErrors(hipFree(u_data));

  // Radix Sort
  checkCudaErrors(hipFree(one_sweep.u_sort));
  checkCudaErrors(hipFree(one_sweep.u_sort_alt));
  checkCudaErrors(hipFree(one_sweep.u_global_histogram));
  checkCudaErrors(hipFree(one_sweep.u_index));
  for (int i = 0; i < radixPasses; ++i) {
    checkCudaErrors(hipFree(one_sweep.u_pass_histograms[i]));
  }

  // Radix Tree
  checkCudaErrors(hipFree(tree.prefixN));
  checkCudaErrors(hipFree(tree.hasLeafLeft));
  checkCudaErrors(hipFree(tree.hasLeafRight));
  checkCudaErrors(hipFree(tree.leftChild));
  checkCudaErrors(hipFree(tree.parent));

  return 0;
}