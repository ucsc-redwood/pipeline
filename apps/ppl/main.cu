#include "hip/hip_runtime.h"
#include <omp.h>

#include <CLI/CLI.hpp>
#include <algorithm>
#include <chrono>
#include <iostream>

#include "cuda/kernels/all.cuh"
#include "kernels/all.hpp"
#include "pipe.cuh"
#include "shared/types.h"

namespace {

struct AppParams {
  int n;
  float min, max, range;
  int seed;
  int my_num_blocks;
};

void k_StdSort(unsigned int* u_data, const int n) {
  std::sort(u_data, u_data + n);
}

// Baseline CPU implementation
void run_cpu_pass(Pipe* pipe, AppParams& params) {
  ++params.seed;
  k_InitRandomVec4(
      pipe->u_points, params.n, params.min, params.range, params.seed);

  k_ComputeMortonCode(pipe->u_points,
                      pipe->one_sweep.u_sort,
                      params.n,
                      params.min,
                      params.range);

  k_SimpleRadixSort(pipe->one_sweep.u_sort, params.n);

  k_Unique(&pipe->n_unique, pipe->one_sweep.u_sort, params.n);
}

// Baseline GPU implementation
void run_gpu_pass(Pipe* pipe, AppParams& params) {
  ++params.seed;

  // Init
  {
    constexpr auto num_threads = 768;
    constexpr auto seed = 114514;
    const auto grid_size = (params.n + num_threads - 1) / num_threads;
    gpu::k_InitRandomVec4<<<grid_size, num_threads>>>(
        pipe->u_points, params.n, params.min, params.range, params.seed);
    // checkCudaErrors(hipDeviceSynchronize());
  }

  // ---------------------------------------------------------------------------
  // Morton Code

  {
    constexpr auto num_threads = 768;

    // int blockSize = 1;
    // int minGridSize = 1;
    // checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
    //     &minGridSize, &blockSize, gpu::k_ComputeMortonCode));
    // spdlog::info("**** blockSize = {}", blockSize);

    gpu::k_ComputeMortonCode<<<params.my_num_blocks, num_threads>>>(
        pipe->u_points,
        pipe->one_sweep.u_sort,
        params.n,
        params.min,
        params.range);
    // checkCudaErrors(hipDeviceSynchronize());
  }

  // gpu::k_InitRandomVec4(
  //     pipe->u_points, params.n, params.min, params.range, params.seed);

  // gpu::k_ComputeMortonCode(pipe->u_points,
  //                          pipe->one_sweep.u_sort,
  //                          params.n,
  //                          params.min,
  //                          params.range);

  // gpu::
  //     // k_SimpleRadixSort(pipe->one_sweep.u_sort, params.n);

  //     k_Unique(&pipe->n_unique, pipe->one_sweep.u_sort, params.n);
}

}  // namespace

int main(const int argc, const char* argv[]) {
  constexpr auto n = 1920 * 1080;  // 2.0736M
  int n_threads = 4;
  int my_num_blocks = 64;

  CLI::App app{"Multi-threaded sorting benchmark"};

  app.add_option("-t,--threads", n_threads, "Number of threads to use")
      ->check(CLI::Range(1, 48));

  app.add_option("-b,--blocks", my_num_blocks, "Number of blocks to use")
      ->check(CLI::PositiveNumber);

  CLI11_PARSE(app, argc, argv)

  spdlog::info("n = {}", n);
  spdlog::info("n_threads = {}", n_threads);
  spdlog::info("my_num_blocks = {}", my_num_blocks);

  omp_set_num_threads(n_threads);

  auto pipe_ptr = std::make_unique<Pipe>(n);

  AppParams params{
      .n = n,
      .min = 0.0f,
      .max = 1024.0f,
      .range = 1024.0f,
      .seed = 114514,
      .my_num_blocks = my_num_blocks,
  };

  constexpr auto n_frames = 100;

  // need to compute the frame rate of this loop
  auto start = std::chrono::steady_clock::now();

  for (auto i = 0; i < n_frames; ++i) {
    run_gpu_pass(pipe_ptr.get(), params);

    // spdlog::info(
    // "[{}/{}] Unique: {}/{}", i, n_frames, pipe_ptr->n_unique, params.n);
    spdlog::info("[{}/{}] ", i, n_frames);
  }
  checkCudaErrors(hipDeviceSynchronize());

  auto end = std::chrono::steady_clock::now();

  auto elapsed_ms =
      std::chrono::duration_cast<std::chrono::milliseconds>(end - start)
          .count();
  double fps = static_cast<double>(n_frames) / (elapsed_ms / 1000.0);

  spdlog::info("Elapsed time: {} ms", elapsed_ms);
  spdlog::info("FPS: {}", fps);

  return 0;
}