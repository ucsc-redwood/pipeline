#include <iostream>
#include "../../benchmarks/common.cuh"

int main(const int argc, const char** argv) {
  int n = 10'000'000;
  auto [d_sort, d_tree] = gpu::MakeRadixTree_Fake();

  

  hipFree(d_sort);
  hipFree(d_tree.prefixN);
  hipFree(d_tree.hasLeafLeft);
  hipFree(d_tree.hasLeafRight);
  hipFree(d_tree.leftChild);
  hipFree(d_tree.parent);
  return 0;
}