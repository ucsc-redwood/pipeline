#include <hip/hip_runtime.h>
#include <omp.h>
#include <spdlog/spdlog.h>

#include <CLI/CLI.hpp>
#include <glm/glm.hpp>

#include "cuda/common.cuh"
#include "cuda/common/helper_cuda.hpp"
#include "kernels/all.hpp"
#include "types/brt.hpp"

int main(const int argc, const char** argv) {
  int n = 10'000'000;
  int n_threads = 4;
  int my_num_blocks = 64;

  CLI::App app{"Multi-threaded sorting benchmark"};

  app.add_option("-n,--n", n, "Number of elements to sort")
      ->check(CLI::PositiveNumber);

  app.add_option("-t,--threads", n_threads, "Number of threads to use")
      ->check(CLI::Range(1, 48));

  app.add_option("-b,--blocks", my_num_blocks, "Number of blocks to use")
      ->check(CLI::PositiveNumber);

  CLI11_PARSE(app, argc, argv)

  spdlog::info("n = {}", n);
  spdlog::info("n_threads = {}", n_threads);

  omp_set_num_threads(n_threads);

  // ---------------------------------------------------------------------------

  unsigned int* u_sort;
  checkCudaErrors(hipMallocManaged(&u_sort, n * sizeof(unsigned int)));

  gpu::k_InitAscendingSync(u_sort, n);

  // peek at the first 10 morton keys
  for (auto i = 0; i < 10; ++i) {
    spdlog::info("u_sort[{}] = {}", i, u_sort[i]);
  }

  const auto num_unique = n;

  // const auto block_size = DetermineBlockSize(gpu::k_BuildRadixTree_Kernel,
  // n); spdlog::info("block_size = {}", block_size);
  // const auto block_size = 256;

  RadixTreeData radix_data;
  radix_data.n_nodes = num_unique - 1;

  // clang-format off
  checkCudaErrors(hipMallocManaged(&radix_data.prefixN, num_unique * sizeof(uint8_t)));
  checkCudaErrors(hipMallocManaged(&radix_data.hasLeafLeft, num_unique * sizeof(bool)));
  checkCudaErrors(hipMallocManaged(&radix_data.hasLeafRight, num_unique * sizeof(bool)));
  checkCudaErrors(hipMallocManaged(&radix_data.leftChild, num_unique * sizeof(int)));
  checkCudaErrors(hipMallocManaged(&radix_data.parent, num_unique * sizeof(int)));
  // clang-format on

  gpu::Dispatch_BuildRadixTree_With(num_unique,
                                    u_sort,
                                    radix_data.prefixN,
                                    radix_data.hasLeafLeft,
                                    radix_data.hasLeafRight,
                                    radix_data.leftChild,
                                    radix_data.parent,
                                    my_num_blocks);
  checkCudaErrors(hipDeviceSynchronize());

  // peek at the first 10 brt nodes

  for (auto i = 0; i < 10; ++i) {
    printf("\n");
    printf("prefixN[%d] = %d\n", i, radix_data.prefixN[i]);
    printf("hasLeafLeft[%d] = %d\n", i, radix_data.hasLeafLeft[i]);
    printf("hasLeafRight[%d] = %d\n", i, radix_data.hasLeafRight[i]);
    printf("leftChild[%d] = %d\n", i, radix_data.leftChild[i]);
    printf("parent[%d] = %d\n", i, radix_data.parent[i]);
  }

  // ---------------------------------------------------------------------------

  checkCudaErrors(hipFree(u_sort));
  checkCudaErrors(hipFree(radix_data.prefixN));
  checkCudaErrors(hipFree(radix_data.hasLeafLeft));
  checkCudaErrors(hipFree(radix_data.hasLeafRight));
  checkCudaErrors(hipFree(radix_data.leftChild));
  checkCudaErrors(hipFree(radix_data.parent));

  return 0;
}